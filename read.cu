#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <sstream>
#include <curl/curl.h>
#include <hip/hip_runtime.h>
#include <cmath>

using namespace std;

// Callback function for writing data received
size_t WriteCallback(void* contents, size_t size, size_t nmemb, void* userp) {
    ((string*)userp)->append((char*)contents, size * nmemb);
    return size * nmemb;
}

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Kernel 1: Reduce Sum Optimized
__global__ void reduceSumOptimized(float* input, float* output, int N) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    sdata[tid] = (i < N ? input[i] : 0) + (i + blockDim.x < N ? input[i + blockDim.x] : 0);
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0) output[blockIdx.x] = sdata[0];
}

// Kernel 2: Compute Mean
__global__ void computeMean(float* sum, float* mean, int size) {
    if (threadIdx.x == 0) {
        *mean = *sum / size;
    }
}

// Kernel 3: Compute Standard Deviation
__global__ void computeStdDev(float* data, float mean, float* stddev, int size) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    float diff = (i < size) ? (data[i] - mean) * (data[i] - mean) : 0;
    sdata[tid] = diff;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0) atomicAdd(stddev, sdata[0]);
}

// Kernel 4: Check Cultivation Feasibility
__global__ void checkFeasibility(float* results, int* feasibilityFlags, int numCultivos, float* limits) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numCultivos) {
        int tempIndex = i * 6;
        feasibilityFlags[i] = (results[0] >= limits[tempIndex] && results[0] <= limits[tempIndex + 1] &&
                               results[1] >= limits[tempIndex + 2] && results[1] <= limits[tempIndex + 3] &&
                               results[2] >= limits[tempIndex + 4] && results[2] <= limits[tempIndex + 5]) ? 1 : 0;
    }
}

// Kernel 5: Compute Viability Index
__global__ void computeViabilityIndex(float* results, float* viabilityIndex, int numCultivos, float* limits) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numCultivos) {
        int tempIndex = i * 6;
        float tempScore = (results[0] - limits[tempIndex]) / (limits[tempIndex + 1] - limits[tempIndex]);
        float humScore = (results[1] - limits[tempIndex + 2]) / (limits[tempIndex + 3] - limits[tempIndex + 2]);
        float luzScore = (results[2] - limits[tempIndex + 4]) / (limits[tempIndex + 5] - limits[tempIndex + 4]);
        viabilityIndex[i] = (tempScore + humScore + luzScore) / 3.0;
    }
}

int main() {
    // Initialize libcurl
    CURL* curl;
    CURLcode res;
    string readBuffer;

    curl = curl_easy_init();
    if (curl) {
        string sheet_id = "1WeLe9zO71zoKhkdj2usptbRQC0Gy6PT6_3CibpJ6gEU";
        string gid = "0";
        string url = "https://docs.google.com/spreadsheets/d/" + sheet_id + "/export?format=csv&gid=" + gid;

        curl_easy_setopt(curl, CURLOPT_URL, url.c_str());
        curl_easy_setopt(curl, CURLOPT_FOLLOWLOCATION, 1L);
        curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, WriteCallback);
        curl_easy_setopt(curl, CURLOPT_WRITEDATA, &readBuffer);

        res = curl_easy_perform(curl);
        curl_easy_cleanup(curl);
    }

    // Parse CSV data
    vector<float> temperatures, humidities, intensities;
    stringstream sstream(readBuffer);
    string line;
    getline(sstream, line);  // Skip header
    while (getline(sstream, line)) {
        stringstream ss(line);
        string value;
        vector<string> row;
        while (getline(ss, value, ',')) row.push_back(value);

        if (row.size() >= 5) {
            temperatures.push_back(stof(row[1]));
            humidities.push_back(stof(row[2]));
            intensities.push_back(stof(row[3]));
        }
    }

    int N = temperatures.size();
    if (N == 0) {
        cout << "No data found." << endl;
        return 1;
    }

    // CUDA configuration
    float *d_data, *d_intermediate, *d_mean, *d_stddev, *d_viabilityIndex;
    vector<float> results(3);  // To store means
    int blockSize = 256;

    for (int i = 0; i < 3; ++i) {
        vector<float>& data = (i == 0 ? temperatures : (i == 1 ? humidities : intensities));
        cudaCheckError(hipMalloc((void**)&d_data, data.size() * sizeof(float)));
        cudaCheckError(hipMemcpy(d_data, data.data(), data.size() * sizeof(float), hipMemcpyHostToDevice));

        int gridSize = (data.size() + blockSize * 2 - 1) / (blockSize * 2);
        cudaCheckError(hipMalloc((void**)&d_intermediate, gridSize * sizeof(float)));
        reduceSumOptimized<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_data, d_intermediate, data.size());
        cudaCheckError(hipDeviceSynchronize());

        // Reduce on CPU
        vector<float> h_intermediate(gridSize);
        cudaCheckError(hipMemcpy(h_intermediate.data(), d_intermediate, gridSize * sizeof(float), hipMemcpyDeviceToHost));
        float totalSum = 0.0f;
        for (float val : h_intermediate) totalSum += val;

        results[i] = round(totalSum / data.size());  // Correct mean calculation

        // Compute standard deviation
        cudaCheckError(hipMalloc((void**)&d_stddev, sizeof(float)));
        cudaCheckError(hipMemset(d_stddev, 0, sizeof(float)));
        computeStdDev<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_data, results[i], d_stddev, data.size());
        cudaCheckError(hipDeviceSynchronize());
        float stddev;
        cudaCheckError(hipMemcpy(&stddev, d_stddev, sizeof(float), hipMemcpyDeviceToHost));
        stddev = sqrt(stddev / data.size());
        cout << "Desviación Estándar (" << (i == 0 ? "Temperatura" : (i == 1 ? "Humedad" : "Intensidad")) << "): " << stddev << endl;

        hipFree(d_data);
        hipFree(d_intermediate);
        hipFree(d_stddev);
    }

    // Crop data
    struct Cultivo {
        string nombre;
        int minTemp, maxTemp, minHum, maxHum, minLuz, maxLuz;
    };

    vector<Cultivo> cultivos = {
        {"Cafe", 18, 24, 60, 80, 40, 60},
        {"Maiz", 20, 30, 50, 70, 50, 100},
        {"Frijol", 15, 25, 50, 70, 60, 80},
        {"Arroz", 24, 35, 70, 90, 60, 80},
        {"Trigo", 10, 25, 30, 50, 60, 80},
    };

    int numCultivos = cultivos.size();
    vector<int> feasibilityFlags(numCultivos);
    vector<float> viabilityIndexes(numCultivos);
    vector<float> limits;

    for (const auto& cultivo : cultivos) {
        limits.push_back(cultivo.minTemp);
        limits.push_back(cultivo.maxTemp);
        limits.push_back(cultivo.minHum);
        limits.push_back(cultivo.maxHum);
        limits.push_back(cultivo.minLuz);
        limits.push_back(cultivo.maxLuz);
    }

    float* d_limits;
    cudaCheckError(hipMalloc((void**)&d_limits, limits.size() * sizeof(float)));
    cudaCheckError(hipMalloc((void**)&d_viabilityIndex, numCultivos * sizeof(float)));
    cudaCheckError(hipMemcpy(d_limits, limits.data(), limits.size() * sizeof(float), hipMemcpyHostToDevice));

    float* d_results;
    int* d_feasibilityFlags;
    cudaCheckError(hipMalloc((void**)&d_results, results.size() * sizeof(float)));
    cudaCheckError(hipMalloc((void**)&d_feasibilityFlags, numCultivos * sizeof(int)));
    cudaCheckError(hipMemcpy(d_results, results.data(), results.size() * sizeof(float), hipMemcpyHostToDevice));

    // Kernel 4: Check feasibility
    checkFeasibility<<<1, numCultivos>>>(d_results, d_feasibilityFlags, numCultivos, d_limits);
    cudaCheckError(hipMemcpy(feasibilityFlags.data(), d_feasibilityFlags, numCultivos * sizeof(int), hipMemcpyDeviceToHost));

    // Kernel 5: Compute viability index
    computeViabilityIndex<<<1, numCultivos>>>(d_results, d_viabilityIndex, numCultivos, d_limits);
    cudaCheckError(hipMemcpy(viabilityIndexes.data(), d_viabilityIndex, numCultivos * sizeof(float), hipMemcpyDeviceToHost));

    vector<string> aptos, noAptos;
    for (int i = 0; i < numCultivos; ++i) {
        if (feasibilityFlags[i]) {
            aptos.push_back(cultivos[i].nombre);
        } else {
            noAptos.push_back(cultivos[i].nombre);
        }
        cout << "Índice de Viabilidad (" << cultivos[i].nombre << "): " << viabilityIndexes[i] << endl;
    }

    // Print final results
    cout << "Media Temperatura: " << results[0] << "°C" << endl;
    cout << "Media Humedad: " << results[1] << "%" << endl;
    cout << "Media Intensidad de Luz: " << results[2] << "%" << endl;

    cout << "Se pueden cultivar: ";
    for (const auto& cultivo : aptos) cout << cultivo << ", ";
    cout << endl;

    cout << "No se pueden cultivar: ";
    for (const auto& cultivo : noAptos) cout << cultivo << ", ";
    cout << endl;

    hipFree(d_limits);
    hipFree(d_results);
    hipFree(d_feasibilityFlags);
    hipFree(d_viabilityIndex);

    return 0;
}
